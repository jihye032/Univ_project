#include "hip/hip_runtime.h"

#include "RayTracingAni.h"
#include "VecMath.h"
#include <stdio.h>
#include <iostream>
#include <math.h>

#define NUM_SPHERE 3

#define BACK 0
#define CEIL 1
#define GROUND 2
#define LEFT 3
#define RIGHT 4

#define M_PI 3.14159265358979323846


// Global Variables
__device__ float3 eye_p;
__device__ Screen screen;
__device__ Sphere spheres[NUM_SPHERE];
__device__ Plane plane[5];
__device__ Cone cone;

__device__ float3 light_source;	// Light Position
__device__ float I;		// Light Intensity



__device__ Ray MakeRay(int pixel_x, int pixel_y)
{
	float3 pixel_center;
	pixel_center.x = screen.min_x + pixel_x*screen.pixel_w + (screen.pixel_w/2.f);
	pixel_center.y = screen.max_y - pixel_y*screen.pixel_h + (screen.pixel_h/2.f);
	pixel_center.z = screen.z;

	Ray r;
	r.p0 = eye_p;

	r.v = Sub(pixel_center, r.p0);
	r.v = Normalize(r.v);

	return r;
}


__global__ void InitEnv(int r_w, int r_h, float elapsed_time)
{
	eye_p = make_float3(0.f, 0.f, -200.f);

	
	screen.w = 300.f;
	screen.h = 200.f;
	screen.min_x = -150.f;
	screen.max_y = 100.f;
	screen.z = -400.f;
	screen.resolution_w = r_w;
	screen.resolution_h = r_h;
	screen.pixel_w = screen.w / r_w;
	screen.pixel_h = screen.h / r_h;

	spheres[0].p = make_float3(0.f, 0.f, -1000.f);
	spheres[0].r = 50.;
	spheres[0].kd = make_float3(0.f, 0.f, 1.f);
	spheres[0].ka = Scale(0.1f, spheres[0].kd);
	spheres[0].ks = 0.7f;
	spheres[0].n = 50.0f;
	spheres[0].reflection = true;

	spheres[1].p = make_float3(sin(elapsed_time*2.)*(-200.f), sin(elapsed_time*2.)*200.f, -1000.f+cos(elapsed_time*1.5f)*300.f);
	spheres[1].r = 50.;
	spheres[1].kd = make_float3(1.f, 0.f, 0.f);
	spheres[1].ka = Scale(0.1f, spheres[1].kd);
	spheres[1].ks = 0.7f;
	spheres[1].n = 50.0f;
	spheres[1].reflection = true;

	spheres[2].p = make_float3(sin(elapsed_time*1.5f)*200.f, sin(elapsed_time*2.)*(-200.f), -1000.f+cos(elapsed_time*2.)*300.f);
	spheres[2].r = 50.;
	spheres[2].kd = make_float3(0.f, 1.f, 0.f);
	spheres[2].ka = Scale(0.1f, spheres[2].kd);
	spheres[2].ks = 0.7f;
	spheres[2].n = 50.0f;
	spheres[2].reflection = true;

	plane[BACK].p = make_float3(0.f, 0.f, -1300.f);
	plane[BACK].N = make_float3(0.f, 0.f, 1.f);
	plane[BACK].kd = make_float3(0.7f, 0.7f, 0.7f);
	plane[BACK].ka = Scale(0.1f, plane[BACK].kd);
	plane[BACK].ks = 0.f;
	plane[BACK].n = 0.0f;
	plane[BACK].reflection = false;

	plane[CEIL].p = make_float3(0.f, 250.f, -1000.f);
	plane[CEIL].N = make_float3(0.f, -1.f, 0.f);
	plane[CEIL].kd = make_float3(0.7f, 0.7f, 0.3f);
	plane[CEIL].ka = Scale(0.1f, plane[CEIL].kd);
	plane[CEIL].ks = 0.f;
	plane[CEIL].n = 0.0f;
	plane[CEIL].reflection = false;

	plane[GROUND].p = make_float3(0.f, -250.f, -1000.f);
	plane[GROUND].N = make_float3(0.f, 1.f, 0.f);
	plane[GROUND].kd = make_float3(0.3f, 0.3f, 0.7f);
	plane[GROUND].ka = Scale(0.1f, plane[GROUND].kd);
	plane[GROUND].ks = 0.f;
	plane[GROUND].n = 0.0f;
	plane[GROUND].reflection = false;

	plane[LEFT].p = make_float3(-400.f, 0.f, -1000.f);
	plane[LEFT].N = make_float3(1.f, 0.f, 0.f);
	plane[LEFT].kd = make_float3(0.7f, 0.3f, 0.3f);
	plane[LEFT].ka = Scale(0.1f, plane[LEFT].kd);
	plane[LEFT].ks = 0.7f;
	plane[LEFT].n = 100.0f;
	plane[LEFT].reflection = true;

	plane[RIGHT].p = make_float3(400.f, 0.f, -1000.f);
	plane[RIGHT].N = make_float3(-1.f, 0.f, 0.f);
	plane[RIGHT].kd = make_float3(0.3f, 0.7f, 0.3f);
	plane[RIGHT].ka = Scale(0.1f, plane[RIGHT].kd);
	plane[RIGHT].ks = 0.7f;
	plane[RIGHT].n = 100.0f;
	plane[RIGHT].reflection = true;


	light_source = make_float3(300.f, 200.f, -650.f);
	I = 1.f;
}

__device__ bool TestIntersection(Ray r, Sphere s, float3 &out_P, float3 &out_N)
{
	float a = 1;
	float b = 2*Dot(r.v, Sub(r.p0, s.p));
	float c = Length2(Sub(r.p0, s.p)) - (s.r*s.r);


	// Determinant Test
	float d = b*b - 4*a*c;
	if ( d < 0 ) return false;

	float t = (-b - sqrt(d))/2.f;

	if ( t < 0.01 ) return false;

	out_P = Add(r.p0, Scale(t, r.v));
	out_N = Normalize( Sub(out_P, s.p) );

	return true;
}

__device__ bool TestIntersection(Ray r, Plane p, float3 &out_P, float3 &out_N)
{
	p.d = -Dot(p.p, p.N);
	if( (Dot(p.p, p.N) + p.d) != 0 ) return false;
	
	float tmp = Dot(r.v, p.N);
	if( tmp == 0 ) return false;

	float t = -(Dot(r.p0, p.N) + p.d) / tmp;
	if( t < 0.01 ) return false;

	//if( Dot(p.N, Sub(r.p0, p.p)) != 0 ) return false;

	out_P = Add( r.p0, Scale(t, r.v) );
	out_N = Normalize( p.N );

	return true;
}

__device__ bool TestIntersection(Ray r, Cone c, float3 &out_P, float3 &out_N)
{
	if ( Length(Sub(r.p0, c.N)) == 0 ) return false;


}

__device__ bool TestIntersection(Ray r, float3 &out_P, float3 &out_N, float3 &out_kd, float3 &out_ka, float &out_ks, float &out_n, float &dis)
{
	bool flag_hit = false;
	float distance = 99999;

	for ( int i=0; i<NUM_SPHERE; i++ )
	{
		float3 tmp_P;
		float3 tmp_N;
		if ( TestIntersection(r, spheres[i], tmp_P, tmp_N) )
		{
			flag_hit = true;
			float tmp_dist = Length(Sub(eye_p, tmp_P));

			if ( distance > tmp_dist )
			{
				dis = Length(Sub(r.p0, tmp_P));

				distance = tmp_dist;
				out_P = tmp_P;
				out_N = tmp_N;
				out_kd = spheres[i].kd;
				out_ka = spheres[i].ka;
				out_ks = spheres[i].ks;
				out_n = spheres[i].n;
			}
		}
	}

	for( int i=0; i<5; i++ )
	{
		float3 tmp_P;
		float3 tmp_N;
		if ( TestIntersection(r, plane[i], tmp_P, tmp_N) )
		{
			float tmp_dist = Length(Sub(eye_p, tmp_P));
			flag_hit = true;

			if ( distance > tmp_dist )
			{
				dis = Length(Sub(r.p0, tmp_P));

				distance = tmp_dist;
				out_P = tmp_P;
				out_N = tmp_N;
				out_kd = plane[i].kd;
				out_ka = plane[i].ka;
				out_ks = plane[i].ks;
				out_n = plane[i].n;

			}
		}
	}

	return flag_hit;
}


__device__ bool TestIntersection(Ray r, float &dis)
{
	float3 P, N;
	float3 kd, ka;
	float ks, n;

	return TestIntersection(r, P, N, kd, ka, ks, n, dis);
}


__device__ float3 DiffuseReflection(float3 L, float3 N, float3 kd)
{
	if ( Dot(N, L) < 0.f) return make_float3(0.f, 0.f, 0.f);
	float3 d = Scale(I*Dot(N, L), kd);
	return d;
}

__device__ float SpecularReflection(float3 L, float3 R, float3 V, float ks, float n)
{
	float s = ks*I*pow(Dot(R, V), n);

	if ( Dot(R, V) < 0 ) return 0;

	return s;
}

__device__ float SpecularReflection2(float3 L, float3 N, float3 P, float ks, float n)
{
	float3 R = Normalize(Sub(Scale(2.f*Dot(L, N), N), L));
	float3 V = Normalize(Sub(eye_p, P));

	return SpecularReflection(L, R, V, ks, n);
}


__device__ bool RayCast(Ray ray, float3& out_color, Ray &out_reflected_ray)
{
	float3 color;

	float3 P, N;
	float3 kd, ka;
	float ks;
	float n, distance;


	if ( TestIntersection(ray, P, N, kd, ka, ks, n, distance) )
	{
		float3 L = Normalize(Sub(light_source, P));
		float3 diffuse = DiffuseReflection(L, N, kd);	//diffuse
		float3 ambient = ka;	//ambient
		float3 specular = Scale(SpecularReflection2(L, N, P, ks, n), make_float3(1.f, 1.f, 1.f));

		
		bool flag_hit = false;

		Ray P_to_L;
		P_to_L.p0 = P;
		P_to_L.v = L;

		float tmp_dis;
		flag_hit = TestIntersection(P_to_L, tmp_dis);

		float LtoP = Length(Sub(light_source, P));

		if ( flag_hit )
		{
			if ( LtoP > tmp_dis )
				color = ambient;

			else
			{
				color = Add(ambient, diffuse);
				color = Add(color, specular);
			}
		}
		else
		{
			color = Add(ambient, diffuse);
			color = Add(color, specular);
		}
		
		out_reflected_ray.p0 = P;
		float3 in_R = Scale(-1.f, ray.v);
		out_reflected_ray.v = Sub(Scale(2*Dot(in_R, N), N), in_R);
	}
	else
	{
		return false;
	}

	out_color = color;


	return true;
}

__device__ bool RayReflection(Ray ray, float3 &out_color, Ray &reflected_ray)
{
	float3 color = make_float3(0.f, 0.f, 0.f);
	for ( int j=0; j<7; j++ )
	{
		float3 tmp_color;
		if ( RayCast(ray, tmp_color, reflected_ray)  )
		{
			ray = reflected_ray;
			color = Add(color, Scale(0.4f, tmp_color));

		}
		else
		{
			break;
		}
	}

	out_color = color;

	return true;
}

__global__ void RayTracing(uchar3* d_color_buffer)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int pixel_x_index = tid%screen.resolution_w;
	int pixel_y_index = tid/screen.resolution_w;
	
	if ( tid < screen.resolution_w*screen.resolution_h )
	{
		
		float3 color;  
		color = make_float3(0.f, 0.f, 0.f);

		Ray ray = MakeRay(pixel_x_index, pixel_y_index);
		Ray next_ray;
		
		for ( int j=0; j<1; j++ )
		{
			float3 tmp_color;
			if ( RayReflection(ray, tmp_color, next_ray) )
			{
				color = Add(color, tmp_color);
				ray = next_ray;
			}
			else
			{
				break;
			}
		}


		// Scale up the color values to 0~255 and Copy the result color
		// R
		int tmp = round(color.x*255);
		if ( tmp < 0 ) tmp = 0;
		if ( tmp > 255 ) tmp = 255;
		d_color_buffer[tid].x = tmp;

		// G
		tmp = round(color.y*255);
		if ( tmp < 0 ) tmp = 0;
		if ( tmp > 255 ) tmp = 255;
		d_color_buffer[tid].y = tmp;

		// B
		tmp = round(color.z*255);
		if ( tmp < 0 ) tmp = 0;
		if ( tmp > 255 ) tmp = 255;
		d_color_buffer[tid].z = tmp;
	}
}


uchar3 *d_color_buffer;	// GPU buffer
int win_resolution_w, win_resolution_h;

void AllocGPUBuffer(int r_w, int r_h)
{
	win_resolution_w = r_w;
	win_resolution_h = r_h;

	hipMalloc(&d_color_buffer, r_w*r_h*sizeof(uchar3));
}

void UpdateImage(uchar3* h_color_buffer, float elapsed_time)
{
    hipError_t cudaStatus;
	int num_rays = win_resolution_w*win_resolution_h;

	InitEnv<<<1, 1>>>(win_resolution_w, win_resolution_h, elapsed_time);
	RayTracing<<<num_rays/512+1, 512>>>(d_color_buffer);

	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

	hipDeviceSynchronize();

	hipMemcpy(h_color_buffer, d_color_buffer, win_resolution_w*win_resolution_h*sizeof(uchar3), hipMemcpyDeviceToHost);
}



void FreeGPUBuffer()
{
	hipFree(d_color_buffer);
}
